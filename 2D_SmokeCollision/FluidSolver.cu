#include "hip/hip_runtime.h"
#include "FluidSolver.cuh"

#define IX(i,j) ((i)+(N+2)*(j))
#define LINEARSOLVERTIMES 10
#define SWAP(x0,x) {double * tmp=x0;x0=x;x=tmp;}

// �ҽ��� �߰� Ŀ��
__global__ void add_source(int N, double* x, double* s, double dt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = (N + 2) * (N + 2);
    if (idx < size)
        x[idx] += dt * s[idx];
}

/* -------------------�׸��� ��� ����------------------- */
// �׸��� �� ��� ���� Ŀ��
__global__ void k_set_bnd(int N, int b, double* x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i <= N) {
        x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
        x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
        x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
        x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
    }
}

// �ڳ� ��� ���� Ŀ��
__global__ void k_update_corners(int N, double* x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i == 0) {
        x[IX(0, 0)] = 0.5 * (x[IX(1, 0)] + x[IX(0, 1)]);
        x[IX(0, N + 1)] = 0.5 * (x[IX(1, N + 1)] + x[IX(0, N)]);
        x[IX(N + 1, 0)] = 0.5 * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
        x[IX(N + 1, N + 1)] = 0.5 * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
    }
}

// �浹 ��� ���� Ŀ��
// 1 : ����, 2 : �ܺ�, 3 : ��, 4 : �Ʒ�, 5 : ������, 6 : ����, 7 : ���� �� �𼭸�, 8 : ������ �� �𼭸�, 9 : ������ �Ʒ� �𼭸�, 10 : ���� �Ʒ� �𼭸�

// �鿡 ���� ��� ����
__global__ void k_collision_bnd(int N, int b, double* x, int* calcResult) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= N && j <= N) {
        int idx = IX(i, j);
        if (calcResult[idx] == 1) {
            x[idx] = 0;
        }
        else if (calcResult[idx] == 3) {
            x[idx] = b == 2 ? -x[IX(i, j + 1)] : x[IX(i, j + 1)];
        }
        else if (calcResult[idx] == 4) {
            x[idx] = b == 2 ? -x[IX(i, j - 1)] : x[IX(i, j - 1)];
        }
        else if (calcResult[idx] == 5) {
            x[idx] = b == 1 ? -x[IX(i + 1, j)] : x[IX(i + 1, j)];
        }
        else if (calcResult[idx] == 6) {
            x[idx] = b == 1 ? -x[IX(i - 1, j)] : x[IX(i - 1, j)];
        }
    }
}

// �ٱ��� �𼭸��� ���� ��� ���� Ŀ��
__global__ void k_collision_outCorner_bnd(int N, double* x, int* calcResult) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= N && j <= N) {
        int idx = IX(i, j);
        if (calcResult[idx] == 7) {
            //x[idx] = 0.5 * (x[IX(i - 1, j)] + x[IX(i, j + 1)]);
            x[idx] = 0.5 * (x[IX(i + 1, j)] + x[IX(i, j - 1)]);
        }
        else if (calcResult[idx] == 8) {
            //x[idx] = 0.5 * (x[IX(i + 1, j)] + x[IX(i, j + 1)]);
            x[idx] = 0.5 * (x[IX(i - 1, j)] + x[IX(i, j - 1)]);
        }
        else if (calcResult[idx] == 9) {
            //x[idx] = 0.5 * (x[IX(i + 1, j)] + x[IX(i, j - 1)]);
            x[idx] = 0.5 * (x[IX(i - 1, j)] + x[IX(i, j + 1)]);
        }
        else if (calcResult[idx] == 10) {
            //x[idx] = 0.5 * (x[IX(i - 1, j)] + x[IX(i, j - 1)]);
            x[idx] = 0.5 * (x[IX(i + 1, j)] + x[IX(i, j + 1)]);
        }
    }
}

// ���� �𼭸��� ���� ��� ���� Ŀ��
__global__ void k_collision_inCorner_bnd(int N, double* x, int* calcResult) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= N && j <= N) {
        int idx = IX(i, j);
        if (calcResult[idx] == 11) {
            x[idx] = 0.5 * (x[IX(i - 1, j)] + x[IX(i, j + 1)]);
        }
        else if (calcResult[idx] == 12) {
            x[idx] = 0.5 * (x[IX(i + 1, j)] + x[IX(i, j + 1)]);
        }
        else if (calcResult[idx] == 13) {
            x[idx] = 0.5 * (x[IX(i + 1, j)] + x[IX(i, j - 1)]);
        }
        else if (calcResult[idx] == 14) {
            x[idx] = 0.5 * (x[IX(i - 1, j)] + x[IX(i, j - 1)]);
        }
    }
}

// �׸��� ��� ���� Ŀ�� ���� �Լ�
void set_bnd(int N, int b, double* x, int* calcIdx)
{
    int blockSize = 256;
    int numBlock = (N + blockSize - 1) / blockSize;
    k_set_bnd << <numBlock, blockSize >> > (N, b, x);

    k_update_corners << <1, 1 >> > (N, x);

    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);
    k_collision_bnd<<<gridDim, blockDim>>>(N, b, x, calcIdx);
    k_collision_outCorner_bnd<<<gridDim, blockDim>>>(N, x, calcIdx);
    k_collision_inCorner_bnd<<<gridDim, blockDim>>>(N, x, calcIdx);
}
/* ----------------------------------------------------- */

/* -----------red black gauss seidel ���� Ǯ��----------- */
// red cell Ŀ��
__global__ void red_cell_lin(int N, double* x, double* x0, double a, double c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i <= N && j <= N && (i + j) % 2 == 0) {
        x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) / c;
    }
}

// black cell Ŀ��
__global__ void black_cell_lin(int N, double* x, double* x0, double a, double c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i <= N && j <= N && (i + j) % 2 == 1) {
        x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) / c;
    }
}

void lin_solve(int N, int b, double* x, double* x0, double a, double c, int* calcIdx) {
    int l;
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    for (l = 0; l < LINEARSOLVERTIMES; l++) {
        red_cell_lin << <gridDim, blockDim >> > (N, x, x0, a, c);
        hipDeviceSynchronize();
        black_cell_lin << <gridDim, blockDim >> > (N, x, x0, a, c);
        hipDeviceSynchronize();
        set_bnd(N, b, x, calcIdx);
        hipDeviceSynchronize();
    }
}
/* ----------------------------------------------------- */

// Ȯ�� �Լ�
void diffuse(int N, int b, double* x, double* x0, double diff, double dt, int* calcIdx)
{
    double a = dt * diff * N * N;
    lin_solve(N, b, x, x0, a, 1 + 4 * a, calcIdx);
}

/* ----------------------�̷� �Լ�---------------------- */
// �̷� Ŀ�� �Լ�
__global__ void k_advect(int N, double* d, double* d0, double* u, double* v, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= N && j <= N) {
        int i0, j0, i1, j1;
        double x, y, s0, t0, s1, t1, dtx, dty;

        dtx = dty = dt * N;
        x = i - dtx * u[IX(i, j)]; y = j - dty * v[IX(i, j)];
        if (x < 0.5f) x = 0.5; if (x > N + 0.5) x = N + 0.5; i0 = (int)x; i1 = i0 + 1;
        if (y < 0.5f) y = 0.5; if (y > N + 0.5) y = N + 0.5; j0 = (int)y; j1 = j0 + 1;

        s1 = x - i0; s0 = 1 - s1; t1 = y - j0; t0 = 1 - t1;

        d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
            s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
    }
}


// �̷� Ŀ�� ���� �Լ�
void advect(int N, int b, double* d, double* d0, double* u, double* v, double dt, int* calcIdx)
{
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    k_advect << <gridDim, blockDim >> > (N, d, d0, u, v, dt);
    hipDeviceSynchronize();

    set_bnd(N, b, d, calcIdx);
    hipDeviceSynchronize();
}
/* ---------------------------------------------------- */

/* ---------------------������Ʈ �Լ�--------------------- */
// �߻��� ����ϰ� �з� �ʵ带 0���� �ʱ�ȭ
__global__ void poison(int N, double* u, double* v, double* p, double* div) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= N && j <= N) {
        double h = 1.0 / N;
        div[IX(i, j)] = -0.5 * h * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]);
        p[IX(i, j)] = 0;
    }
}

// �ӵ� �ʵ� ������Ʈ (���� ����)
__global__ void k_project(int N, double* u, double* v, double* p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= N && j <= N) {
        double h = 1.0 / N;
        u[IX(i, j)] -= 0.5 * (p[IX(i + 1, j)] - p[IX(i - 1, j)]) / h;
        v[IX(i, j)] -= 0.5 * (p[IX(i, j + 1)] - p[IX(i, j - 1)]) / h;
    }
}

// ������Ʈ Ŀ�� �Լ��� �����ϴ� �Լ�
void project(int N, double* u, double* v, double* p, double* div, int* calcIdx)
{
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);
    poison<<<gridDim, blockDim>>>(N, u, v, p, div);
    hipDeviceSynchronize();

    set_bnd(N, 0, div, calcIdx); set_bnd(N, 0, p, calcIdx);
    hipDeviceSynchronize();

    lin_solve(N, 0, p, div, 1, 4, calcIdx);

    k_project<<<gridDim, blockDim>>>(N, u, v, p);
    hipDeviceSynchronize();

    set_bnd(N, 1, u, calcIdx); set_bnd(N, 2, v, calcIdx);
    hipDeviceSynchronize();
}
/* ------------------------------------------------------ */

// �е� �ʵ� ������Ʈ
void dens_step(int N, double* x, double* x0, double* u, double* v, double diff, double dt, int* calcIdx)
{
    // add source kernel
    int size = (N + 2) * (N + 2);
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    add_source<<<numBlocks, blockSize>>>(N, x, x0, dt);
    hipDeviceSynchronize();

    // swap and diffuse
    SWAP(x0, x); diffuse(N, 0, x, x0, diff, dt, calcIdx);
    SWAP(x0, x); advect(N, 0, x, x0, u, v, dt, calcIdx);
}

// �ӵ� �ʵ� ������Ʈ
void vel_step(int N, double* u, double* v, double* u0, double* v0, double visc, double dt, int* calcIdx)
{
    // add source kernel
    int size = (N + 2) * (N + 2);
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    add_source<<<numBlocks, blockSize>>>(N, u, u0, dt); add_source<<<numBlocks, blockSize>>>(N, v, v0, dt);
    hipDeviceSynchronize();

    // swap and diffuse
    SWAP(u0, u); diffuse(N, 1, u, u0, visc, dt, calcIdx);
    SWAP(v0, v); diffuse(N, 2, v, v0, visc, dt, calcIdx);

    // project and swap
    project(N, u, v, u0, v0, calcIdx);
    SWAP(u0, u); SWAP(v0, v);

    // advect
    advect(N, 1, u, u0, u0, v0, dt, calcIdx); advect(N, 2, v, v0, u0, v0, dt, calcIdx);

    // final project
    project(N, u, v, u0, v0, calcIdx);
}