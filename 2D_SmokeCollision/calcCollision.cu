#include "hip/hip_runtime.h"
#include "calcCollision.cuh"

#define IX(i, j) ((i) + (N+2)*(j))
#define DIX(i, j) ((i) + (N)*(j))

calcCollision::calcCollision(int N, double dx, double dy, float scale) {
	init(N, dx, dy, scale);
}

calcCollision::~calcCollision() {
	hipFree(draw_InsideCell);
	hipFree(draw_OutsideCell);
	hipFree(collisionResult_D);
	hipFree(calc_InsideCell);
	hipFree(calc_OutsideCell);
	hipFree(collisionResult_IX);
}

void calcCollision::init(int N, double dx, double dy, float scale) {
	cx = dx;
	cy = dy;
	cScale = scale;

	hipMalloc((void**)&draw_InsideCell, N * N * sizeof(int));
	hipMalloc((void**)&draw_OutsideCell, N * N * sizeof(int));
	hipMalloc((void**)&collisionResult_D, N  * N * sizeof(int));
	hipMalloc((void**)&calc_InsideCell, (N + 2) * (N + 2) * sizeof(int));
	hipMalloc((void**)&calc_OutsideCell, (N + 2) * (N + 2) * sizeof(int));
	hipMalloc((void**)&collisionResult_IX, (N + 2) * (N + 2) * sizeof(int));

	hipMemset(draw_InsideCell, 0, N * N * sizeof(int));
	hipMemset(draw_OutsideCell, 0, N * N * sizeof(int));
	hipMemset(collisionResult_D, 0, N * N * sizeof(int));
	hipMemset(calc_InsideCell, 0, (N + 2) * (N + 2) * sizeof(int));
	hipMemset(calc_OutsideCell, 0, (N + 2) * (N + 2) * sizeof(int));
	hipMemset(collisionResult_IX, 0, (N + 2) * (N + 2) * sizeof(int));
}

__global__ void collision_kernel(int N, glm::vec3 sphere_center, float sphere_radius, int* drawResult, int* calcResult, double dx, double dy) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < N && j < N) {
		int didx = DIX(i, j);
		int idx = IX(i + 1, j + 1);
		double h, x, y;
		h = 1.0 / N;
		x = (i - 0.5) * h + dx;
		y = (j - 0.5) * h + dy;

		glm::vec3 cell_center(x, y, 0.0f);

		// ��ü�� �� �߽��� ���� �Ÿ� ���
		float distance = glm::length(cell_center - sphere_center);

		// �浹 ����
		if (distance <= sphere_radius) {
			drawResult[didx] = 1;  // �浹 �߻�
			calcResult[idx] = 1;
		}
		else {
			drawResult[didx] = 0;  // �浹 ����
			calcResult[idx] = 0;
		}
	}
}

__global__ void divide_collision_draw(int N, int* insideCell, int* outsideCell, int* drawResult) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= 1 && i < N - 1 && j >= 1 && j < N - 1) {  // �迭 ��踦 ����� �ʵ��� ����
		int idx = DIX(i, j);
		if (drawResult[idx] == 1) {
			if (drawResult[DIX(i - 1, j)] == 0 || drawResult[DIX(i + 1, j)] == 0 ||
				drawResult[DIX(i, j - 1)] == 0 || drawResult[DIX(i, j + 1)] == 0) {
				drawResult[idx] = 2;  // �ܺ� ��
			}
			else {
				drawResult[idx] = 1;  // ���� ��
			}
		}
		else {
			drawResult[idx] = 0;  // �浹�� �Ͼ�� ���� ��
		}
	}
}

void calcCollision::check_collision(int N) {
	glm::vec3 cameraPos = getCameraPosition();
	glm::vec3 cameraFront = getCameraDirection();
	float t = -cameraPos.z / cameraFront.z;
	glm::vec3 pointInWorld = cameraPos + t * cameraFront;
	double xpos = pointInWorld.x;
	double ypos = pointInWorld.y;

	dim3 blockDim(16, 16);
	dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

	glm::vec3 sphere_center(xpos, ypos, 0.0f);
	collision_kernel<<<gridDim, blockDim>>>(N, sphere_center, cScale, collisionResult_D, collisionResult_IX, cx, cy);
	hipDeviceSynchronize();

	divide_collision_draw<<<gridDim, blockDim>>>(N, draw_InsideCell, draw_OutsideCell, collisionResult_D);
}