#include "hip/hip_runtime.h"
#include "drawSphere.cuh"
#include "control.h"

#define DIX(i, j) ((i) + (N) * (j))
#define M_PI 3.141592

drawSphere::drawSphere(int N) {
	init(N);
}

drawSphere::~drawSphere() {
	delete sphereColors;
	glDeleteBuffers(1, &spherebuffer);
	glDeleteBuffers(1, &spereColorBuffer);
	hipFree(d_collision_result);
	hipGraphicsUnregisterResource(cudaVBOsphere);
}

__global__ void init_sphere(int stacks, int slices, glm::vec3* sphere, double x, double y, double scale) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < stacks && j < slices) {
		float stackInterval = M_PI / (float)stacks;
		float sliceInterval = 2.0 * M_PI / (float)slices;

		float stackAngle1 = i * stackInterval;
		float stackAngle2 = (i + 1) * stackInterval;

		float sliceAngle1 = j * sliceInterval;
		float sliceAngle2 = (j + 1) * sliceInterval;
			
		glm::vec3 vertex1 = glm::vec3(
			x + scale * sinf(stackAngle1) * cosf(sliceAngle1),
			y + scale * cosf(stackAngle1),
			scale * sinf(stackAngle1) * sinf(sliceAngle1)
		);

		glm::vec3 vertex2 = glm::vec3(
			x + scale * sinf(stackAngle2) * cosf(sliceAngle1),
			y + scale * cosf(stackAngle2),
			scale * sinf(stackAngle2) * sinf(sliceAngle1)
		);

		glm::vec3 vertex3 = glm::vec3(
			x + scale * sinf(stackAngle1) * cosf(sliceAngle2),
			y + scale * cosf(stackAngle1),
			scale * sinf(stackAngle1) * sinf(sliceAngle2)
		);

		glm::vec3 vertex4 = glm::vec3(
			x + scale * sinf(stackAngle2) * cosf(sliceAngle2),
			y + scale * cosf(stackAngle2),
			scale * sinf(stackAngle2) * sinf(sliceAngle2)
		);

		int index = (i * slices + j) * 6;
		sphere[index + 0] = vertex1;
		sphere[index + 1] = vertex2;
		sphere[index + 2] = vertex3;

		sphere[index + 3] = vertex2;
		sphere[index + 4] = vertex4;
		sphere[index + 5] = vertex3;
	}
}

void drawSphere::init_camera() {
	// ��ü ���� ��ġ
	glm::vec3 cameraPos = getCameraPosition();
	glm::vec3 cameraFront = getCameraDirection();
	float t = -cameraPos.z / cameraFront.z;
	glm::vec3 pointInWorld = cameraPos + t * cameraFront;
	xpos = pointInWorld.x;
	ypos = pointInWorld.y;
}

void drawSphere::init(int N) {
	dim3 blockDim(16, 16);
	dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

	numStacks = 20;
	numSlices = 20;
	sphereNum = 6 * numStacks * numSlices;
	sphereScale = 0.05;

	init_camera();

	// ��ü ����
	glGenBuffers(1, &spherebuffer);
	glBindBuffer(GL_ARRAY_BUFFER, spherebuffer);
	glBufferData(GL_ARRAY_BUFFER, sphereNum * sizeof(glm::vec3), NULL, GL_STATIC_DRAW);

	hipGraphicsGLRegisterBuffer(&cudaVBOsphere, spherebuffer, cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsMapResources(1, &cudaVBOsphere, 0);
	hipGraphicsResourceGetMappedPointer((void**)&d_sphere_buffer, &numBytesphere, cudaVBOsphere);
	init_sphere<<<gridDim, blockDim>>>(numStacks, numSlices, d_sphere_buffer, xpos, ypos, sphereScale);
	hipDeviceSynchronize();
	hipGraphicsUnmapResources(1, &cudaVBOsphere, 0);

	sphereColors = new glm::vec3[sphereNum];
	for (int i = 0; i < sphereNum; ++i) {
		sphereColors[i] = glm::vec3(0.0f, 1.0f, 0.0f);
	}

	glGenBuffers(1, &spereColorBuffer);
	glBindBuffer(GL_ARRAY_BUFFER, spereColorBuffer);
	glBufferData(GL_ARRAY_BUFFER, sphereNum * sizeof(glm::vec3), sphereColors, GL_STATIC_DRAW);

	// �浹 ���� ����
	hipMalloc((void**)&d_collision_result, N * N * sizeof(int));
	hipMemset(d_collision_result, 0, N * N * sizeof(int));
}

__global__ void check_collision(int N, glm::vec3 sphere_center, float sphere_radius, int* collision_result, double dx, double dy) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < N && j < N) {
		int idx = DIX(i, j);
		double h, x, y;
		h = 1.0 / N;
		x = (i - 0.5) * h + dx;
		y = (j - 0.5) * h + dy;

		glm::vec3 cell_center(x, y, 0.0f);

		// ��ü�� �� �߽��� ���� �Ÿ� ���
		float distance = glm::length(cell_center - sphere_center);

		// �浹 ����
		if (distance <= sphere_radius) {
			collision_result[idx] = 1;  // �浹 �߻�
		}
		else {
			collision_result[idx] = 0;  // �浹 ����
		}
	}
}


void drawSphere::drawSph(int N, double dx, double dy) {
	glm::vec3 cameraPos = getCameraPosition();
	glm::vec3 cameraFront = getCameraDirection();
	float t = -cameraPos.z / cameraFront.z;
	glm::vec3 pointInWorld = cameraPos + t * cameraFront;
	xpos = pointInWorld.x;
	ypos = pointInWorld.y;

	dim3 blockDim(16, 16);
	dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

	glm::vec3 sphere_center(xpos, ypos, 0.0f);
	check_collision<<<gridDim, blockDim>>>(N, sphere_center, sphereScale, d_collision_result, dx, dy);

	hipGraphicsMapResources(1, &cudaVBOsphere, 0);
	hipGraphicsResourceGetMappedPointer((void**)&d_sphere_buffer, &numBytesphere, cudaVBOsphere);
	init_sphere << <gridDim, blockDim >> > (numStacks, numSlices, d_sphere_buffer, xpos, ypos, sphereScale);
	hipGraphicsUnmapResources(1, &cudaVBOsphere, 0);

	glBindBuffer(GL_ARRAY_BUFFER, spherebuffer);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(
		0,
		3,
		GL_FLOAT,
		GL_FALSE,
		0,
		(void*)0
	);

	glBindBuffer(GL_ARRAY_BUFFER, spereColorBuffer);
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(
		1,
		3,
		GL_FLOAT,
		GL_FALSE,
		0,
		(void*)0
	);
	glDrawArrays(GL_TRIANGLES, 0, sphereNum);
	glDisableVertexAttribArray(0);
	glDisableVertexAttribArray(1);
}