#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <glm/glm.hpp>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "control.h"
#include "shader.h"
#include "FluidSolver.cuh"
#include "velocity.cuh"
#include "density.cuh"
#include "drawSphere.cuh"

velocity* _vel;
density* _den;
drawSphere* _sphere;

// 2���� �ε����� 1���� �ε���ó�� ����
#define IX(i, j) ((i) + (N+2)*(j))
#define DIX(i, j) ((i) + (N)*(j))

// �׸��� ũ��
#define SIZE 300

// ȭ�� ũ��
#define WINDOW_WIDTH 800
#define WINDOW_HEIGHT 800

GLFWwindow* window;

// �ֹ��� ���� GPU �޸� �Ҵ� ����
static double* u, * v, * u_prev, * v_prev;
static double* dens, * dens_prev;

// �ֹ��� ���� ��� ������
static const int N = SIZE;
static double dt = 0.08;
static double diff = 0.0;
static double visc = 0.0;
static double force = 10.0;
static double source = 50.0f;

// �ùķ��̼� ���� ����
static int addforce[3] = { 0, 0, 0 };
static int mode = 0;

static int width = WINDOW_WIDTH;
static int height = WINDOW_HEIGHT;

// ������ �Ҹ�
void free_data() {
	if (u) hipFree(u);
	if (v) hipFree(v);
	if (u_prev) hipFree(u_prev);
	if (v_prev) hipFree(v_prev);
	if (dens) hipFree(dens);
	if (dens_prev) hipFree(dens_prev);
}

/* --------------------������ �ʱ�ȭ-------------------- */
// ������ �ʱⰪ ���� Ŀ�� �Լ�
__global__ void initArray(double* array, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		array[i] = 0.0;
	}
}

// �ʱ�ȭ Ŀ�� ���� �Լ�
static void init_data() {
	int size = (N + 2) * (N + 2);
	size_t d_size = size * sizeof(double);

	hipMalloc((void**)&u, d_size);
	hipMalloc((void**)&v, d_size);
	hipMalloc((void**)&u_prev, d_size);
	hipMalloc((void**)&v_prev, d_size);
	hipMalloc((void**)&dens, d_size);
	hipMalloc((void**)&dens_prev, d_size);

	int blockSize = 256;
	int numBlocks = (size + blockSize - 1) / blockSize;
	initArray<<<numBlocks, blockSize>>>(u, size);
	initArray<<<numBlocks, blockSize>>>(v, size);
	initArray<<<numBlocks, blockSize>>>(u_prev, size);
	initArray<<<numBlocks, blockSize>>>(v_prev, size);
	initArray<<<numBlocks, blockSize>>>(dens, size);
	initArray<<<numBlocks, blockSize>>>(dens_prev, size);
}
/* ---------------------------------------------------- */

/* ------------------�ҽ��� �߰� �Լ�------------------ */
__global__ void setForceAndSource(double* d, double* v, int i1, int j1, double forceValue, int i2, int j2, double sourceValue) {
	v[IX(i1, j1)] = forceValue;
	d[IX(i2, j2)] = sourceValue;
}

void get_force_source(double* d, double* u, double* v) {
	int i, j, size = (N + 2) * (N + 2);
	hipMemset(u, 0, size * sizeof(double));
	hipMemset(v, 0, size * sizeof(double));
	hipMemset(d, 0, size * sizeof(double));

	double forceValue;
	double sourceValue;

	if (addforce[0] == 1) {
		i = N / 2;
		j = 2;

		if (i < 1 || i > N || j < 1 || j > N) {
			std::cerr << "���� ���" << '\n';
			return;
		}

		forceValue = force * 3;
		sourceValue = source;
		setForceAndSource<<<1, 1>>>(d, v, i, j, forceValue, i, 10, sourceValue);
	}
}
/* --------------------------------------------------- */

// �ùķ��̼� ���� �Լ�
void sim_fluid() {
	get_force_source(dens_prev, u_prev, v_prev);
	vel_step(N, u, v, u_prev, v_prev, visc, dt);
	dens_step(N, dens, dens_prev, u, v, diff, dt);
	hipDeviceSynchronize();
}

// Ű���� �ݹ� �Լ�
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods) {
	if (key == GLFW_KEY_Z && action == GLFW_RELEASE) {
		addforce[0] = (addforce[0] == 0) ? 1 : 0;
		std::cout << "addforce[0] : " << addforce[0] << '\n';
	}

	if (key == GLFW_KEY_X && action == GLFW_RELEASE) {
		addforce[1] = (addforce[1] == 0) ? 1 : 0;
		std::cout << "addforce[1] : " << addforce[1] << '\n';
	}

	if (key == GLFW_KEY_1 && action == GLFW_RELEASE) {
		mode = 0;
		std::cout << "mode : " << mode << '\n';
	}

	if (key == GLFW_KEY_2 && action == GLFW_RELEASE) {
		mode = 1;
		std::cout << "mode : " << mode << '\n';
	}
}

int main() {
	// GLFW �ʱ�ȭ
	if (!glfwInit()) {
		std::cerr << "GLFW �ʱ�ȭ ����" << '\n';
		glfwTerminate();
		return -1;
	}
	glfwWindowHint(GLFW_SAMPLES, 4);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 5);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	window = glfwCreateWindow(width, height, "collision test", NULL, NULL);
	if (window == NULL) {
		std::cerr << "GLFW �ʱ�ȭ ����" << '\n';
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	glewExperimental = true;
	if (glewInit() != GLEW_OK) {
		std::cerr << "GLFW �ʱ�ȭ ����" << '\n';
		glfwTerminate();
		return -1;
	}

	// ���� �ʱ�ȭ
	init_data();
	hipDeviceSynchronize();

	// �ӵ�, �е� Ŭ���� �ʱ�ȭ
	double drawX = -0.5f;
	double drawY = -0.5f;
	_vel = new velocity(N, drawX, drawY);
	_den = new density(N, drawX, drawY);
	_sphere = new drawSphere(N);

	// ���̴� �б�
	GLuint programID = LoadShaders("VertexShaderSL.txt", "FragmentShaderSL.txt");
	GLuint MatrixID = glGetUniformLocation(programID, "MVP");
	GLuint alpValue = glGetUniformLocation(programID, "alphaValue");

	// ���콺 ����
	glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
	glfwPollEvents();
	glfwSetCursorPos(window, width / 2, height / 2);

	hipSetDevice(0);

	GLuint VertexArrayID;
	glGenVertexArrays(1, &VertexArrayID);
	glBindVertexArray(VertexArrayID);


	glfwSetKeyCallback(window, key_callback);
	glfwSetInputMode(window, GLFW_STICKY_KEYS, GL_TRUE);
	glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	do {
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

		glUseProgram(programID);

		// ȭ�� �̵�, ��Ʈ�� control.h
		computeMatricesFromInputs(window, width, height);
		glm::mat4 ProjectionMatrix = getProjectionMatrix();
		glm::mat4 ViewMatrix = getViewMatrix();
		glm::mat4 ModelMatrix = glm::mat4(1.0);
		glm::mat4 MVP = ProjectionMatrix * ViewMatrix * ModelMatrix;
		glUniformMatrix4fv(MatrixID, 1, GL_FALSE, &MVP[0][0]);

		// �ùķ��̼� �ݺ�
		sim_fluid();

		glUniform1f(alpValue, 1.0f);
		if (mode == 0) {
			_den->draw_dens(N, dens);
		}
		if (mode == 1) {
			_vel->draw_velocity(N, u, v);
		}

		glUniform1f(alpValue, 0.3f);
		_sphere->drawSph(N);

		glfwSwapBuffers(window);
		glfwPollEvents();
	} while ((glfwGetKey(window, GLFW_KEY_ESCAPE) != GLFW_PRESS && glfwWindowShouldClose(window) == 0));

	// ������ ����
	glDeleteProgram(programID);
	glDeleteVertexArrays(1, &VertexArrayID);
	glfwDestroyWindow(window);
	free_data();
	delete _vel;
	glfwTerminate();

	return 0;
}